#include "timer.h"

namespace lfm {
void GPUTimer::Start(void)
{
    hipEventRecord(start);
}

void GPUTimer::Stop()
{
    hipEventRecord(stop);
}

void GPUTimer::Elapsed(const std::string& _message)
{
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("%s: %f ms\n", _message.c_str(), time);
}
}