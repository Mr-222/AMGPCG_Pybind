#include "util.h"
#include <iostream>

namespace lfm {
template <typename T>
void DevToDevCpyAsync(T* _dst, const T* _src, int _size, hipStream_t _stream)
{
    hipMemcpyAsync((void*)_dst, (const void*)_src, _size * sizeof(T), hipMemcpyDeviceToDevice, _stream);
}

template void DevToDevCpyAsync<int>(int*, const int*, int, hipStream_t);
template void DevToDevCpyAsync<float>(float*, const float*, int, hipStream_t);
};