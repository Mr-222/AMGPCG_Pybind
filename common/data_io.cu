#include "hip/hip_runtime.h"
#include "data_io.h"
#include <algorithm>
#include <array>
#include <complex>
#include <cstdint>
#include <cstring>
#include <fstream>
#include <iostream>
#include <iterator>
#include <sstream>
#include <stdexcept>
#include <string>
#include <type_traits>
#include <typeindex>
#include <typeinfo>
#include <unordered_map>
#include <utility>
#include <vector>

namespace lfm {
template <typename T>
__global__ void ConToTileKernel(T* _dst, int3 _tile_dim, const T* _src)
{
    int tile_idx  = blockIdx.x;
    int3 tile_ijk = TileIdxToIjk(_tile_dim, tile_idx);
    int t_id      = threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int voxel_idx  = i * 128 + t_id;
        int3 voxel_ijk = VoxelIdxToIjk(voxel_idx);
        int dst_idx    = tile_idx * 512 + voxel_idx;
        int src_idx    = (tile_ijk.x * 8 + voxel_ijk.x) * _tile_dim.y * _tile_dim.z * 64 + (tile_ijk.y * 8 + voxel_ijk.y) * _tile_dim.z * 8 + tile_ijk.z * 8 + voxel_ijk.z;
        _dst[dst_idx]  = _src[src_idx];
    }
}

template <typename T>
void ConToTileAsync(DHMemory<T>& _dst, int3 _tile_dim, const DHMemory<T>& _src, hipStream_t _stream)
{
    T* dst       = _dst.dev_ptr_;
    const T* src = _src.dev_ptr_;
    int tile_num = Prod(_tile_dim);
    ConToTileKernel<<<tile_num, 128, 0, _stream>>>(dst, _tile_dim, src);
}

template <typename T>
__global__ void TileToConKernel(T* _dst, int3 _tile_dim, const T* _src)
{
    int tile_idx  = blockIdx.x;
    int3 tile_ijk = TileIdxToIjk(_tile_dim, tile_idx);
    int t_id      = threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int voxel_idx  = i * 128 + t_id;
        int3 voxel_ijk = VoxelIdxToIjk(voxel_idx);
        int src_idx    = tile_idx * 512 + voxel_idx;
        int dst_idx    = (tile_ijk.x * 8 + voxel_ijk.x) * _tile_dim.y * _tile_dim.z * 64 + (tile_ijk.y * 8 + voxel_ijk.y) * _tile_dim.z * 8 + tile_ijk.z * 8 + voxel_ijk.z;
        _dst[dst_idx]  = _src[src_idx];
    }
}

template <typename T>
void TileToConAsync(DHMemory<T>& _dst, int3 _tile_dim, const DHMemory<T>& _src, hipStream_t _stream)
{
    T* dst       = _dst.dev_ptr_;
    const T* src = _src.dev_ptr_;
    int tile_num = Prod(_tile_dim);
    TileToConKernel<<<tile_num, 128, 0, _stream>>>(dst, _tile_dim, src);
}

template <typename T>
__global__ void StagConToTileXKernel(T* _dst_x, int3 _tile_dim, const T* _src_x)
{
    int tile_idx    = blockIdx.x;
    int3 x_tile_dim = { _tile_dim.x + 1, _tile_dim.y, _tile_dim.z };
    int3 tile_ijk   = TileIdxToIjk(x_tile_dim, tile_idx);
    int3 x_max_ijk  = { _tile_dim.x * 8, _tile_dim.y * 8 - 1, _tile_dim.z * 8 - 1 };
    int t_id        = threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int voxel_idx  = i * 128 + t_id;
        int3 voxel_ijk = VoxelIdxToIjk(voxel_idx);
        int3 ijk       = { tile_ijk.x * 8 + voxel_ijk.x, tile_ijk.y * 8 + voxel_ijk.y, tile_ijk.z * 8 + voxel_ijk.z };
        int dst_idx    = tile_idx * 512 + voxel_idx;
        if (ijk.x <= x_max_ijk.x && ijk.y <= x_max_ijk.y && ijk.z <= x_max_ijk.z) {
            int y_dim       = _tile_dim.y * 8;
            int z_dim       = _tile_dim.z * 8;
            int src_idx     = ijk.x * y_dim * z_dim + ijk.y * z_dim + ijk.z;
            _dst_x[dst_idx] = _src_x[src_idx];
        } else
            _dst_x[dst_idx] = 0;
    }
}

template <typename T>
__global__ void StagConToTileYKernel(T* _dst_y, int3 _tile_dim, const T* _src_y)
{
    int tile_idx    = blockIdx.x;
    int3 y_tile_dim = { _tile_dim.x, _tile_dim.y + 1, _tile_dim.z };
    int3 tile_ijk   = TileIdxToIjk(y_tile_dim, tile_idx);
    int3 y_max_ijk  = { _tile_dim.x * 8 - 1, _tile_dim.y * 8, _tile_dim.z * 8 - 1 };
    int t_id        = threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int voxel_idx  = i * 128 + t_id;
        int3 voxel_ijk = VoxelIdxToIjk(voxel_idx);
        int3 ijk       = { tile_ijk.x * 8 + voxel_ijk.x, tile_ijk.y * 8 + voxel_ijk.y, tile_ijk.z * 8 + voxel_ijk.z };
        int dst_idx    = tile_idx * 512 + voxel_idx;
        if (ijk.x <= y_max_ijk.x && ijk.y <= y_max_ijk.y && ijk.z <= y_max_ijk.z) {
            int y_dim       = _tile_dim.y * 8 + 1;
            int z_dim       = _tile_dim.z * 8;
            int src_idx     = ijk.x * y_dim * z_dim + ijk.y * z_dim + ijk.z;
            _dst_y[dst_idx] = _src_y[src_idx];
        } else
            _dst_y[dst_idx] = 0;
    }
}

template <typename T>
__global__ void StagConToTileZKernel(T* _dst_z, int3 _tile_dim, const T* _src_z)
{
    int tile_idx    = blockIdx.x;
    int3 z_tile_dim = { _tile_dim.x, _tile_dim.y, _tile_dim.z + 1 };
    int3 tile_ijk   = TileIdxToIjk(z_tile_dim, tile_idx);
    int3 z_max_ijk  = { _tile_dim.x * 8 - 1, _tile_dim.y * 8 - 1, _tile_dim.z * 8 };
    int t_id        = threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int voxel_idx  = i * 128 + t_id;
        int3 voxel_ijk = VoxelIdxToIjk(voxel_idx);
        int3 ijk       = { tile_ijk.x * 8 + voxel_ijk.x, tile_ijk.y * 8 + voxel_ijk.y, tile_ijk.z * 8 + voxel_ijk.z };
        int dst_idx    = tile_idx * 512 + voxel_idx;
        if (ijk.x <= z_max_ijk.x && ijk.y <= z_max_ijk.y && ijk.z <= z_max_ijk.z) {
            int y_dim       = _tile_dim.y * 8;
            int z_dim       = _tile_dim.z * 8 + 1;
            int src_idx     = ijk.x * y_dim * z_dim + ijk.y * z_dim + ijk.z;
            _dst_z[dst_idx] = _src_z[src_idx];
        } else
            _dst_z[dst_idx] = 0;
    }
}

template <typename T>
void StagConToTileAsync(DHMemory<T>& _dst_x, DHMemory<T>& _dst_y, DHMemory<T>& _dst_z, int3 _tile_dim, const DHMemory<T>& _src_x, const DHMemory<T>& _src_y, const DHMemory<T>& _src_z, hipStream_t _stream)
{
    int3 x_tile_dim = { _tile_dim.x + 1, _tile_dim.y, _tile_dim.z };
    int3 y_tile_dim = { _tile_dim.x, _tile_dim.y + 1, _tile_dim.z };
    int3 z_tile_dim = { _tile_dim.x, _tile_dim.y, _tile_dim.z + 1 };
    T* dst_x        = _dst_x.dev_ptr_;
    T* dst_y        = _dst_y.dev_ptr_;
    T* dst_z        = _dst_z.dev_ptr_;
    const T* src_x  = _src_x.dev_ptr_;
    const T* src_y  = _src_y.dev_ptr_;
    const T* src_z  = _src_z.dev_ptr_;
    StagConToTileXKernel<<<Prod(x_tile_dim), 128, 0, _stream>>>(dst_x, _tile_dim, src_x);
    StagConToTileYKernel<<<Prod(y_tile_dim), 128, 0, _stream>>>(dst_y, _tile_dim, src_y);
    StagConToTileZKernel<<<Prod(z_tile_dim), 128, 0, _stream>>>(dst_z, _tile_dim, src_z);
}

template <typename T>
__global__ void StagTileToConXKernel(T* _dst_x, int3 _tile_dim, const T* _src_x)
{
    int tile_idx    = blockIdx.x;
    int3 x_tile_dim = { _tile_dim.x + 1, _tile_dim.y, _tile_dim.z };
    int3 tile_ijk   = TileIdxToIjk(x_tile_dim, tile_idx);
    int3 x_max_ijk  = { _tile_dim.x * 8, _tile_dim.y * 8 - 1, _tile_dim.z * 8 - 1 };
    int t_id        = threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int voxel_idx  = i * 128 + t_id;
        int3 voxel_ijk = VoxelIdxToIjk(voxel_idx);
        int3 ijk       = { tile_ijk.x * 8 + voxel_ijk.x, tile_ijk.y * 8 + voxel_ijk.y, tile_ijk.z * 8 + voxel_ijk.z };
        int src_idx    = tile_idx * 512 + voxel_idx;
        if (ijk.x <= x_max_ijk.x && ijk.y <= x_max_ijk.y && ijk.z <= x_max_ijk.z) {
            int y_dim       = _tile_dim.y * 8;
            int z_dim       = _tile_dim.z * 8;
            int dst_idx     = ijk.x * y_dim * z_dim + ijk.y * z_dim + ijk.z;
            _dst_x[dst_idx] = _src_x[src_idx];
        }
    }
}

template <typename T>
__global__ void StagTileToConYKernel(T* _dst_y, int3 _tile_dim, const T* _src_y)
{
    int tile_idx    = blockIdx.x;
    int3 y_tile_dim = { _tile_dim.x, _tile_dim.y + 1, _tile_dim.z };
    int3 tile_ijk   = TileIdxToIjk(y_tile_dim, tile_idx);
    int3 y_max_ijk  = { _tile_dim.x * 8 - 1, _tile_dim.y * 8, _tile_dim.z * 8 - 1 };
    int t_id        = threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int voxel_idx  = i * 128 + t_id;
        int3 voxel_ijk = VoxelIdxToIjk(voxel_idx);
        int3 ijk       = { tile_ijk.x * 8 + voxel_ijk.x, tile_ijk.y * 8 + voxel_ijk.y, tile_ijk.z * 8 + voxel_ijk.z };
        int src_idx    = tile_idx * 512 + voxel_idx;
        if (ijk.x <= y_max_ijk.x && ijk.y <= y_max_ijk.y && ijk.z <= y_max_ijk.z) {
            int y_dim       = _tile_dim.y * 8 + 1;
            int z_dim       = _tile_dim.z * 8;
            int dst_idx     = ijk.x * y_dim * z_dim + ijk.y * z_dim + ijk.z;
            _dst_y[dst_idx] = _src_y[src_idx];
        }
    }
}

template <typename T>
__global__ void StagTileToConZKernel(T* _dst_z, int3 _tile_dim, const T* _src_z)
{
    int tile_idx    = blockIdx.x;
    int3 z_tile_dim = { _tile_dim.x, _tile_dim.y, _tile_dim.z + 1 };
    int3 tile_ijk   = TileIdxToIjk(z_tile_dim, tile_idx);
    int3 z_max_ijk  = { _tile_dim.x * 8 - 1, _tile_dim.y * 8 - 1, _tile_dim.z * 8 };
    int t_id        = threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int voxel_idx  = i * 128 + t_id;
        int3 voxel_ijk = VoxelIdxToIjk(voxel_idx);
        int3 ijk       = { tile_ijk.x * 8 + voxel_ijk.x, tile_ijk.y * 8 + voxel_ijk.y, tile_ijk.z * 8 + voxel_ijk.z };
        int src_idx    = tile_idx * 512 + voxel_idx;
        if (ijk.x <= z_max_ijk.x && ijk.y <= z_max_ijk.y && ijk.z <= z_max_ijk.z) {
            int y_dim       = _tile_dim.y * 8;
            int z_dim       = _tile_dim.z * 8 + 1;
            int dst_idx     = ijk.x * y_dim * z_dim + ijk.y * z_dim + ijk.z;
            _dst_z[dst_idx] = _src_z[src_idx];
        }
    }
}

template <typename T>
void StagTileToConAsync(DHMemory<T>& _dst_x, DHMemory<T>& _dst_y, DHMemory<T>& _dst_z, int3 _tile_dim, const DHMemory<T>& _src_x, const DHMemory<T>& _src_y, const DHMemory<T>& _src_z, hipStream_t _stream)
{
    int3 x_tile_dim = { _tile_dim.x + 1, _tile_dim.y, _tile_dim.z };
    int3 y_tile_dim = { _tile_dim.x, _tile_dim.y + 1, _tile_dim.z };
    int3 z_tile_dim = { _tile_dim.x, _tile_dim.y, _tile_dim.z + 1 };
    T* dst_x        = _dst_x.dev_ptr_;
    T* dst_y        = _dst_y.dev_ptr_;
    T* dst_z        = _dst_z.dev_ptr_;
    const T* src_x  = _src_x.dev_ptr_;
    const T* src_y  = _src_y.dev_ptr_;
    const T* src_z  = _src_z.dev_ptr_;
    StagTileToConXKernel<<<Prod(x_tile_dim), 128, 0, _stream>>>(dst_x, _tile_dim, src_x);
    StagTileToConYKernel<<<Prod(y_tile_dim), 128, 0, _stream>>>(dst_y, _tile_dim, src_y);
    StagTileToConZKernel<<<Prod(z_tile_dim), 128, 0, _stream>>>(dst_z, _tile_dim, src_z);
}

namespace npy {
    /*
    Copyright 2017-2023 Leon Merten Lohse

    Permission is hereby granted, free of charge, to any person obtaining a copy
    of this software and associated documentation files (the "Software"), to deal
    in the Software without restriction, including without limitation the rights
    to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
    copies of the Software, and to permit persons to whom the Software is
    furnished to do so, subject to the following conditions:

    The above copyright notice and this permission notice shall be included in
    all copies or substantial portions of the Software.

    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
    AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
    OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
    SOFTWARE.
    */

    /* Compile-time test for byte order.
       If your compiler does not define these per default, you may want to define
       one of these constants manually.
       Defaults to little endian order. */
#if defined(__BYTE_ORDER) && __BYTE_ORDER == __BIG_ENDIAN || defined(__BIG_ENDIAN__) || defined(__ARMEB__) || defined(__THUMBEB__) || defined(__AARCH64EB__) || defined(_MIBSEB) || defined(__MIBSEB) || defined(__MIBSEB__)
    const bool big_endian = true;
#else
    const bool big_endian = false;
#endif

    const size_t magic_string_length                         = 6;
    const std::array<char, magic_string_length> magic_string = { '\x93', 'N', 'U', 'M', 'P', 'Y' };

    const char little_endian_char = '<';
    const char big_endian_char    = '>';
    const char no_endian_char     = '|';

    constexpr std::array<char, 3> endian_chars  = { little_endian_char, big_endian_char, no_endian_char };
    constexpr std::array<char, 4> numtype_chars = { 'f', 'i', 'u', 'c' };

    constexpr char host_endian_char = (big_endian ? big_endian_char : little_endian_char);

    /* npy array length */
    using ndarray_len_t = unsigned long int;
    using shape_t       = std::vector<ndarray_len_t>;

    using version_t = std::pair<char, char>;

    struct dtype_t {
        char byteorder;
        char kind;
        unsigned int itemsize;

        inline std::string str() const
        {
            std::stringstream ss;
            ss << byteorder << kind << itemsize;
            return ss.str();
        }

        inline std::tuple<const char, const char, const unsigned int> tie() const
        {
            return std::tie(byteorder, kind, itemsize);
        }
    };

    struct header_t {
        dtype_t dtype;
        bool fortran_order;
        shape_t shape;
    };

    inline void write_magic(std::ostream& ostream, version_t version)
    {
        ostream.write(magic_string.data(), magic_string_length);
        ostream.put(version.first);
        ostream.put(version.second);
    }

    inline version_t read_magic(std::istream& istream)
    {
        std::array<char, magic_string_length + 2> buf {};
        istream.read(buf.data(), sizeof(buf));

        if (!istream) {
            throw std::runtime_error("io error: failed reading file");
        }

        if (!std::equal(magic_string.begin(), magic_string.end(), buf.begin()))
            throw std::runtime_error("this file does not have a valid npy format.");

        version_t version;
        version.first  = buf[magic_string_length];
        version.second = buf[magic_string_length + 1];

        return version;
    }

    const std::unordered_map<std::type_index, dtype_t> dtype_map = {
        { std::type_index(typeid(float)), { host_endian_char, 'f', sizeof(float) } },
        { std::type_index(typeid(double)), { host_endian_char, 'f', sizeof(double) } },
        { std::type_index(typeid(long double)), { host_endian_char, 'f', sizeof(long double) } },
        { std::type_index(typeid(char)), { no_endian_char, 'i', sizeof(char) } },
        { std::type_index(typeid(signed char)), { no_endian_char, 'i', sizeof(signed char) } },
        { std::type_index(typeid(short)), { host_endian_char, 'i', sizeof(short) } },
        { std::type_index(typeid(int)), { host_endian_char, 'i', sizeof(int) } },
        { std::type_index(typeid(long)), { host_endian_char, 'i', sizeof(long) } },
        { std::type_index(typeid(long long)), { host_endian_char, 'i', sizeof(long long) } },
        { std::type_index(typeid(unsigned char)), { no_endian_char, 'u', sizeof(unsigned char) } },
        { std::type_index(typeid(unsigned short)), { host_endian_char, 'u', sizeof(unsigned short) } },
        { std::type_index(typeid(unsigned int)), { host_endian_char, 'u', sizeof(unsigned int) } },
        { std::type_index(typeid(unsigned long)), { host_endian_char, 'u', sizeof(unsigned long) } },
        { std::type_index(typeid(unsigned long long)), { host_endian_char, 'u', sizeof(unsigned long long) } },
        { std::type_index(typeid(std::complex<float>)), { host_endian_char, 'c', sizeof(std::complex<float>) } },
        { std::type_index(typeid(std::complex<double>)), { host_endian_char, 'c', sizeof(std::complex<double>) } },
        { std::type_index(typeid(std::complex<long double>)), { host_endian_char, 'c', sizeof(std::complex<long double>) } }
    };

    // helpers
    inline bool is_digits(const std::string& str) { return std::all_of(str.begin(), str.end(), ::isdigit); }

    template <typename T, size_t N>
    inline bool in_array(T val, const std::array<T, N>& arr)
    {
        return std::find(std::begin(arr), std::end(arr), val) != std::end(arr);
    }

    inline dtype_t parse_descr(std::string typestring)
    {
        if (typestring.length() < 3) {
            throw std::runtime_error("invalid typestring (length)");
        }

        char byteorder_c       = typestring.at(0);
        char kind_c            = typestring.at(1);
        std::string itemsize_s = typestring.substr(2);

        if (!in_array(byteorder_c, endian_chars)) {
            throw std::runtime_error("invalid typestring (byteorder)");
        }

        if (!in_array(kind_c, numtype_chars)) {
            throw std::runtime_error("invalid typestring (kind)");
        }

        if (!is_digits(itemsize_s)) {
            throw std::runtime_error("invalid typestring (itemsize)");
        }
        unsigned int itemsize = std::stoul(itemsize_s);

        return { byteorder_c, kind_c, itemsize };
    }

    namespace pyparse {

        /**
          Removes leading and trailing whitespaces
          */
        inline std::string trim(const std::string& str)
        {
            const std::string whitespace = " \t";
            auto begin                   = str.find_first_not_of(whitespace);

            if (begin == std::string::npos)
                return "";

            auto end = str.find_last_not_of(whitespace);

            return str.substr(begin, end - begin + 1);
        }

        inline std::string get_value_from_map(const std::string& mapstr)
        {
            size_t sep_pos = mapstr.find_first_of(":");
            if (sep_pos == std::string::npos)
                return "";

            std::string tmp = mapstr.substr(sep_pos + 1);
            return trim(tmp);
        }

        /**
           Parses the string representation of a Python dict

           The keys need to be known and may not appear anywhere else in the data.
         */
        inline std::unordered_map<std::string, std::string> parse_dict(std::string in, const std::vector<std::string>& keys)
        {
            std::unordered_map<std::string, std::string> map;

            if (keys.size() == 0)
                return map;

            in = trim(in);

            // unwrap dictionary
            if ((in.front() == '{') && (in.back() == '}'))
                in = in.substr(1, in.length() - 2);
            else
                throw std::runtime_error("Not a Python dictionary.");

            std::vector<std::pair<size_t, std::string>> positions;

            for (auto const& value : keys) {
                size_t pos = in.find("'" + value + "'");

                if (pos == std::string::npos)
                    throw std::runtime_error("Missing '" + value + "' key.");

                std::pair<size_t, std::string> position_pair { pos, value };
                positions.push_back(position_pair);
            }

            // sort by position in dict
            std::sort(positions.begin(), positions.end());

            for (size_t i = 0; i < positions.size(); ++i) {
                std::string raw_value;
                size_t begin { positions[i].first };
                size_t end { std::string::npos };

                std::string key = positions[i].second;

                if (i + 1 < positions.size())
                    end = positions[i + 1].first;

                raw_value = in.substr(begin, end - begin);

                raw_value = trim(raw_value);

                if (raw_value.back() == ',')
                    raw_value.pop_back();

                map[key] = get_value_from_map(raw_value);
            }

            return map;
        }

        /**
          Parses the string representation of a Python boolean
          */
        inline bool parse_bool(const std::string& in)
        {
            if (in == "True")
                return true;
            if (in == "False")
                return false;

            throw std::runtime_error("Invalid python boolan.");
        }

        /**
          Parses the string representation of a Python str
          */
        inline std::string parse_str(const std::string& in)
        {
            if ((in.front() == '\'') && (in.back() == '\''))
                return in.substr(1, in.length() - 2);

            throw std::runtime_error("Invalid python string.");
        }

        /**
          Parses the string represenatation of a Python tuple into a vector of its items
         */
        inline std::vector<std::string> parse_tuple(std::string in)
        {
            std::vector<std::string> v;
            const char seperator = ',';

            in = trim(in);

            if ((in.front() == '(') && (in.back() == ')'))
                in = in.substr(1, in.length() - 2);
            else
                throw std::runtime_error("Invalid Python tuple.");

            std::istringstream iss(in);

            for (std::string token; std::getline(iss, token, seperator);) {
                v.push_back(token);
            }

            return v;
        }

        template <typename T>
        inline std::string write_tuple(const std::vector<T>& v)
        {
            if (v.size() == 0)
                return "()";

            std::ostringstream ss;
            ss.imbue(std::locale("C"));

            if (v.size() == 1) {
                ss << "(" << v.front() << ",)";
            } else {
                const std::string delimiter = ", ";
                // v.size() > 1
                ss << "(";
                std::copy(v.begin(), v.end() - 1, std::ostream_iterator<T>(ss, delimiter.c_str()));
                ss << v.back();
                ss << ")";
            }

            return ss.str();
        }

        inline std::string write_boolean(bool b)
        {
            if (b)
                return "True";
            else
                return "False";
        }

    } // namespace pyparse

    inline header_t parse_header(std::string header)
    {
        /*
           The first 6 bytes are a magic string: exactly "x93NUMPY".
           The next 1 byte is an unsigned byte: the major version number of the file
           format, e.g. x01. The next 1 byte is an unsigned byte: the minor version
           number of the file format, e.g. x00. Note: the version of the file format
           is not tied to the version of the numpy package. The next 2 bytes form a
           little-endian unsigned short int: the length of the header data HEADER_LEN.
           The next HEADER_LEN bytes form the header data describing the array's
           format. It is an ASCII string which contains a Python literal expression of
           a dictionary. It is terminated by a newline ('n') and padded with spaces
           ('x20') to make the total length of the magic string + 4 + HEADER_LEN be
           evenly divisible by 16 for alignment purposes. The dictionary contains
           three keys:

           "descr" : dtype.descr
           An object that can be passed as an argument to the numpy.dtype()
           constructor to create the array's dtype. "fortran_order" : bool Whether the
           array data is Fortran-contiguous or not. Since Fortran-contiguous arrays
           are a common form of non-C-contiguity, we allow them to be written directly
           to disk for efficiency. "shape" : tuple of int The shape of the array. For
           repeatability and readability, this dictionary is formatted using
           pprint.pformat() so the keys are in alphabetic order.
         */

        // remove trailing newline
        if (header.back() != '\n')
            throw std::runtime_error("invalid header");
        header.pop_back();

        // parse the dictionary
        std::vector<std::string> keys { "descr", "fortran_order", "shape" };
        auto dict_map = npy::pyparse::parse_dict(header, keys);

        if (dict_map.size() == 0)
            throw std::runtime_error("invalid dictionary in header");

        std::string descr_s   = dict_map["descr"];
        std::string fortran_s = dict_map["fortran_order"];
        std::string shape_s   = dict_map["shape"];

        std::string descr = npy::pyparse::parse_str(descr_s);
        dtype_t dtype     = parse_descr(descr);

        // convert literal Python bool to C++ bool
        bool fortran_order = npy::pyparse::parse_bool(fortran_s);

        // parse the shape tuple
        auto shape_v = npy::pyparse::parse_tuple(shape_s);

        shape_t shape;
        for (auto item : shape_v) {
            auto dim = static_cast<ndarray_len_t>(std::stoul(item));
            shape.push_back(dim);
        }

        return { dtype, fortran_order, shape };
    }

    inline std::string write_header_dict(const std::string& descr, bool fortran_order, const shape_t& shape)
    {
        std::string s_fortran_order = npy::pyparse::write_boolean(fortran_order);
        std::string shape_s         = npy::pyparse::write_tuple(shape);

        return "{'descr': '" + descr + "', 'fortran_order': " + s_fortran_order + ", 'shape': " + shape_s + ", }";
    }

    inline void write_header(std::ostream& out, const header_t& header)
    {
        std::string header_dict = write_header_dict(header.dtype.str(), header.fortran_order, header.shape);

        size_t length = magic_string_length + 2 + 2 + header_dict.length() + 1;

        version_t version { 1, 0 };
        if (length >= 255 * 255) {
            length  = magic_string_length + 2 + 4 + header_dict.length() + 1;
            version = { 2, 0 };
        }
        size_t padding_len = 16 - length % 16;
        std::string padding(padding_len, ' ');

        // write magic
        write_magic(out, version);

        // write header length
        if (version == version_t { 1, 0 }) {
            auto header_len = static_cast<uint16_t>(header_dict.length() + padding.length() + 1);

            std::array<uint8_t, 2> header_len_le16 { static_cast<uint8_t>((header_len >> 0) & 0xff),
                                                     static_cast<uint8_t>((header_len >> 8) & 0xff) };
            out.write(reinterpret_cast<char*>(header_len_le16.data()), 2);
        } else {
            auto header_len = static_cast<uint32_t>(header_dict.length() + padding.length() + 1);

            std::array<uint8_t, 4> header_len_le32 {
                static_cast<uint8_t>((header_len >> 0) & 0xff), static_cast<uint8_t>((header_len >> 8) & 0xff),
                static_cast<uint8_t>((header_len >> 16) & 0xff), static_cast<uint8_t>((header_len >> 24) & 0xff)
            };
            out.write(reinterpret_cast<char*>(header_len_le32.data()), 4);
        }

        out << header_dict << padding << '\n';
    }

    inline std::string read_header(std::istream& istream)
    {
        // check magic bytes an version number
        version_t version = read_magic(istream);

        uint32_t header_length = 0;
        if (version == version_t { 1, 0 }) {
            std::array<uint8_t, 2> header_len_le16 {};
            istream.read(reinterpret_cast<char*>(header_len_le16.data()), 2);
            header_length = (header_len_le16[0] << 0) | (header_len_le16[1] << 8);

            if ((magic_string_length + 2 + 2 + header_length) % 16 != 0) {
                // TODO(llohse): display warning
            }
        } else if (version == version_t { 2, 0 }) {
            std::array<uint8_t, 4> header_len_le32 {};
            istream.read(reinterpret_cast<char*>(header_len_le32.data()), 4);

            header_length = (header_len_le32[0] << 0) | (header_len_le32[1] << 8) | (header_len_le32[2] << 16) | (header_len_le32[3] << 24);

            if ((magic_string_length + 2 + 4 + header_length) % 16 != 0) {
                // TODO(llohse): display warning
            }
        } else {
            throw std::runtime_error("unsupported file format version");
        }

        auto buf_v = std::vector<char>(header_length);
        istream.read(buf_v.data(), header_length);
        std::string header(buf_v.data(), header_length);

        return header;
    }

    inline ndarray_len_t comp_size(const shape_t& shape)
    {
        ndarray_len_t size = 1;
        for (ndarray_len_t i : shape)
            size *= i;

        return size;
    }

    template <typename Scalar>
    struct npy_data {
        std::vector<Scalar> data = {};
        shape_t shape            = {};
        bool fortran_order       = false;
    };

    template <typename Scalar>
    struct npy_data_ptr {
        const Scalar* data_ptr = nullptr;
        shape_t shape          = {};
        bool fortran_order     = false;
    };

    template <typename Scalar>
    inline npy_data<Scalar> read_npy(std::istream& in)
    {
        std::string header_s = read_header(in);

        // parse header
        header_t header = parse_header(header_s);

        // check if the typestring matches the given one
        const dtype_t dtype = dtype_map.at(std::type_index(typeid(Scalar)));

        if (header.dtype.tie() != dtype.tie()) {
            throw std::runtime_error("formatting error: typestrings not matching");
        }

        // compute the data size based on the shape
        auto size = static_cast<size_t>(comp_size(header.shape));

        npy_data<Scalar> data;

        data.shape         = header.shape;
        data.fortran_order = header.fortran_order;

        data.data.resize(size);

        // read the data
        in.read(reinterpret_cast<char*>(data.data.data()), sizeof(Scalar) * size);

        return data;
    }

    template <typename Scalar>
    inline npy_data<Scalar> read_npy(const std::string& filename)
    {
        std::ifstream stream(filename, std::ifstream::binary);
        if (!stream) {
            throw std::runtime_error("io error: failed to open a file.");
        }

        return read_npy<Scalar>(stream);
    }

    template <typename Scalar>
    inline void write_npy(std::ostream& out, const npy_data<Scalar>& data)
    {
        //  static_assert(has_typestring<Scalar>::value, "scalar type not
        //  understood");
        const dtype_t dtype = dtype_map.at(std::type_index(typeid(Scalar)));

        header_t header { dtype, data.fortran_order, data.shape };
        write_header(out, header);

        auto size = static_cast<size_t>(comp_size(data.shape));

        out.write(reinterpret_cast<const char*>(data.data.data()), sizeof(Scalar) * size);
    }

    template <typename Scalar>
    inline void write_npy(const std::string& filename, const npy_data<Scalar>& data)
    {
        std::ofstream stream(filename, std::ofstream::binary);
        if (!stream) {
            throw std::runtime_error("io error: failed to open a file.");
        }

        write_npy<Scalar>(stream, data);
    }

    template <typename Scalar>
    inline void write_npy(std::ostream& out, const npy_data_ptr<Scalar>& data_ptr)
    {
        const dtype_t dtype = dtype_map.at(std::type_index(typeid(Scalar)));

        header_t header { dtype, data_ptr.fortran_order, data_ptr.shape };
        write_header(out, header);

        auto size = static_cast<size_t>(comp_size(data_ptr.shape));

        out.write(reinterpret_cast<const char*>(data_ptr.data_ptr), sizeof(Scalar) * size);
    }

    template <typename Scalar>
    inline void write_npy(const std::string& filename, const npy_data_ptr<Scalar>& data_ptr)
    {
        std::ofstream stream(filename, std::ofstream::binary);
        if (!stream) {
            throw std::runtime_error("io error: failed to open a file.");
        }

        write_npy<Scalar>(stream, data_ptr);
    }

    // old interface

    // NOLINTBEGIN(*-avoid-c-arrays)
    template <typename Scalar>
    inline void SaveArrayAsNumpy(const std::string& filename, bool fortran_order, unsigned int n_dims,
                                 const unsigned long shape[], const Scalar* data)
    {
        const npy_data_ptr<Scalar> ptr { data, { shape, shape + n_dims }, fortran_order };

        write_npy<Scalar>(filename, ptr);
    }

    template <typename Scalar>
    inline void SaveArrayAsNumpy(const std::string& filename, bool fortran_order, unsigned int n_dims,
                                 const unsigned long shape[], const std::vector<Scalar>& data)
    {
        SaveArrayAsNumpy(filename, fortran_order, n_dims, shape, data.data());
    }

    template <typename Scalar>
    inline void LoadArrayFromNumpy(const std::string& filename, std::vector<unsigned long>& shape, bool& fortran_order,
                                   std::vector<Scalar>& data)
    {
        const npy_data<Scalar> n_data = read_npy<Scalar>(filename);

        shape         = n_data.shape;
        fortran_order = n_data.fortran_order;

        std::copy(n_data.data.begin(), n_data.data.end(), std::back_inserter(data));
    }

    template <typename Scalar>
    inline void LoadArrayFromNumpy(const std::string& filename, std::vector<unsigned long>& shape,
                                   std::vector<Scalar>& data)
    {
        bool fortran_order = false;
        LoadArrayFromNumpy<Scalar>(filename, shape, fortran_order, data);
    }
    // NOLINTEND(*-avoid-c-arrays)

}; // namespace npy

template <typename T>
void ReadNpy(std::string _file, T* _data)
{
    npy::npy_data<T> d = npy::read_npy<T>(_file);
    int size           = 1;
    for (int i = 0; i < d.shape.size(); i++)
        size *= d.shape[i];
    for (int i = 0; i < size; i++)
        _data[i] = d.data[i];
}

template <typename T>
void WriteNpy(std::string _file, int3 _grid_dim, const T* _data)
{
    npy::npy_data<T> d;
    int size = Prod(_grid_dim);
    d.data.resize(size);
    d.shape         = { (unsigned long int)(_grid_dim.x), (unsigned long int)(_grid_dim.y), (unsigned long int)(_grid_dim.z) };
    d.fortran_order = false;
    for (int i = 0; i < size; i++)
        d.data[i] = _data[i];
    npy::write_npy<T>(_file, d);
}

template void TileToConAsync<uint8_t>(DHMemory<uint8_t>&, int3, const DHMemory<uint8_t>&, hipStream_t);
template void TileToConAsync<float>(DHMemory<float>&, int3, const DHMemory<float>&, hipStream_t);
template void ConToTileAsync<uint8_t>(DHMemory<uint8_t>&, int3, const DHMemory<uint8_t>&, hipStream_t);
template void ConToTileAsync<float>(DHMemory<float>&, int3, const DHMemory<float>&, hipStream_t);
template void StagTileToConAsync<float>(DHMemory<float>&, DHMemory<float>&, DHMemory<float>&, int3, const DHMemory<float>&, const DHMemory<float>&, const DHMemory<float>&, hipStream_t);
template void StagConToTileAsync<float>(DHMemory<float>&, DHMemory<float>&, DHMemory<float>&, int3, const DHMemory<float>&, const DHMemory<float>&, const DHMemory<float>&, hipStream_t);

template void ReadNpy<char>(std::string, char*);
template void ReadNpy<float>(std::string, float*);
template void WriteNpy<char>(std::string, int3, const char*);
template void WriteNpy<float>(std::string, int3, const float*);
};